#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CHECK(call) \
do { \
    hipError_t status = call; \
    if (status != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d: %s\n", \
            __FILE__, __LINE__, hipGetErrorString(status)); \
        exit(1); \
    } \
} while (0)

__global__ void fill_array(short2 *arr, int size, int tile_size)
{
    __shared__ short2 tile[16][16];

    int tile_x = blockIdx.x * tile_size;
    int tile_y = blockIdx.y * tile_size;
    int i_start = tile_x + threadIdx.x;
    int j_start = tile_y + threadIdx.y;

    for (int i = i_start; i < tile_x + tile_size && i < size; i += blockDim.x)
    {
        for (int j = j_start; j < tile_y + tile_size && j < size; j += blockDim.y)
        {
            tile[threadIdx.x][threadIdx.y] = make_short2(i,j);
            arr[i*size + j] = tile[threadIdx.x][threadIdx.y];
        }
    }
}

int main()
{
    int size = 40000;
    int tile_size = 64;
    short2 *d_arr, *h_arr;
    size_t arr_size = size * size * sizeof(short2);

    // Allocate pinned memory on the CPU
    CUDA_CHECK(hipHostMalloc((void**)&h_arr, arr_size, hipHostMallocDefault));

    // Allocate memory on the GPU
    CUDA_CHECK(hipMalloc((void**)&d_arr, arr_size));

    // Launch the kernel to fill the array
    dim3 block_size(16, 16);
    dim3 grid_size((size + tile_size - 1) / tile_size, (size + tile_size - 1) / tile_size);
    for (int tile_x = 0; tile_x < size; tile_x += tile_size)
    {
        for (int tile_y = 0; tile_y < size; tile_y += tile_size)
        {
            dim3 tile_grid_size((tile_size + block_size.x - 1) / block_size.x, (tile_size + block_size.y - 1) / block_size.y);
            fill_array<<<tile_grid_size, block_size>>>(d_arr + tile_x * size + tile_y, size, tile_size);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
        }
    }

    // Copy the result back to the CPU
    CUDA_CHECK(hipMemcpy(h_arr, d_arr, arr_size, hipMemcpyDeviceToHost));

    // Print the last element of the array
    printf("%d %d\n", h_arr[size * size - 1].x, h_arr[size * size - 1].y);

    // Free the memory
    CUDA_CHECK(hipFree(d_arr));
    free(h_arr);
    return 0;
}
